/*
 * To compile: nvcc maxgpu.cu
 */
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

unsigned int getmax(unsigned int *, unsigned int);
__global__ void getmaxcu(unsigned int num[], unsigned int size);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    unsigned int * numbers_d; //pointer to the numbers array on the device

    if(argc !=2)
    {
        printf("usage: maxseq num\n");
        printf("num = size of the array\n");
        exit(1);
    }

    size = atol(argv[1]);
    //int numBlocks = ceil(size/1024);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    numbers_d = (unsigned int *)malloc(size * sizeof(unsigned int));

    if( !numbers )
    {
        printf("Unable to allocate mem for an array of size %u\n", size);
        exit(1);
    }

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1
    for( i = 0; i < size; i++)
        numbers[i] = rand() % size;

    //printf(" The maximum number in the array is: %u\n",
    //       getmax(numbers, size));

    /*
     * 1. allocate device memory
     * 2. copy numbers array to device
     * 3. each SM finds local max
     * 4. write back to host
     */
    hipMalloc((void**)&numbers_d, size* sizeof(unsigned int));
    hipMemcpy(numbers_d, numbers, size* sizeof(unsigned int), hipMemcpyHostToDevice);

    getmaxcu<<<1, 1024>>>(numbers_d, size);

    hipMemcpy(&numbers[0], &numbers_d[0], sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(numbers_d);

    printf("The maximum number in the array is %u\n", numbers[0]);
    free(numbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmax(unsigned int num[], unsigned int size)
{
    unsigned int i;
    unsigned int max = num[0];

    for(i = 1; i < size; i++)
        if(num[i] > max)
            max = num[i];

    return( max );
}
/*
 * Find max in own section of the array, keep updating max.
 */
__global__ void getmaxcu(unsigned int num[], unsigned int size){
    int i = blockIdx.x *blockDim.x + threadIdx.x;
    int stop = i + size/1024;

    for(i; i < stop; i++){
        if(num[i] > num[0])
            num[0] = num[i];
    }
}
